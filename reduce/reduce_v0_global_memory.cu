#include <hip/hip_runtime.h>

#include <iostream>

#define THREAD_PER_BLOCK 256            // 每个block有256个线程
#define N 32 * 1024 * 1024              // 共N个元素进行规约求和


__global__ void reduce(float *d_input, float *d_output)
{
    // 1、设计block
    // 3 1 7 0 4 1 6 3
    // 4   7   5   9
    // 11      14
    // 25
    // 2、为每个block设计一个起始索引  每个Block的第一个元素都是input_begin[0]
    float *input_begin = d_input + blockIdx.x * blockDim.x;
    // 3、写代码要从从线程角度思考
    // if (threadIdx.x == 0 or 2 or 4 or 6)
    //     input_begin[threadIdx.x] += input_begin[threadIdx.x + 1];
    // if (threadIdx.x == 0 or 4)
    //     input_begin[threadIdx.x] += input_begin[threadIdx.x + 2];
    // if (threadIdx.x == 0)
    //     input_begin[threadIdx.x] += input_begin[threadIdx.x + 4];
    // 4、如果线程不是8个，而是很多个呢？所以不能写成if  应该写成for
    // for (int i = 1; i < blockDim.x; i *= 2)
    // {
    //     if (threadIdx.x % (i * 2) == 0)
    //     {
    //         input_begin[threadIdx.x] += input_begin[threadIdx.x + i];
    //     }    
    // }
    // 5、同步问题：3+1计算完了，但是如果7+0还没计算完，就计算4+7 那么肯定会出问题
    for (int i = 1; i < blockDim.x; i *= 2)
    {
        if (threadIdx.x % (i * 2) == 0)
        {
            input_begin[threadIdx.x] += input_begin[threadIdx.x + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        // d_output跟block数量相同
        d_output[blockIdx.x] = input_begin[0];
    }
}

bool check(float *h_result, float *h_output, int n)
{
    for (int i = 0; i < n; i++)
    {
        if(abs(h_result[i] - h_output[i]) > 1e-4)
        {
            return false;
        }
    }
    return true;
}

int main()
{
    // std::cout << "Hello Reduce!" << std::endl;
    float *h_input = (float *)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK;
    float *h_output = (float *)malloc((block_num) * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, (block_num) * sizeof(float));

    float *h_result = (float *)malloc((block_num) * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        // h_input[i] = (float)i;z
        h_input[i] = 2.0 * (float)drand48() - 1.0;
    }

    // cpu上计算
    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < THREAD_PER_BLOCK; j++)
        {
            cur += h_input[i * THREAD_PER_BLOCK + j];
        }
        h_result[i] = cur;
    }

    // gpu上计算
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);
    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);
    reduce<<<Grid, Block>>>(d_input, d_output);
    hipMemcpy(h_output, d_output, (block_num) * sizeof(float), hipMemcpyDeviceToHost);

    // check result
    if (check(h_result, h_output, block_num))
    {
        std::cout << "Success" << std::endl;
    } else 
    {
        for (int i = 0; i < block_num; ++i)
        {
            std::cout << h_output[i] << " ";
        }
        std::cout << std::endl;
        std::cout << "Fail" << std::endl;
    }

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}